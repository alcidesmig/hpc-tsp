#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>
#include <time.h>

#define ll long long
#define MAX_THREAD_PER_BLOCK 1024
#define MAX_NB_CITIES 50
#define DEBUG

__device__ int calc_perm_cost(ll idx, int n, ll nb_perm, int * dist, ll * fact, int tasks_per_thread, ll limit_blocks_1d) {
	// Test for valid idx
	int cont_tasks = 0, cost = 0;
	int min_cost = INT_MAX;
	// Resulting perm vector
	int perm[MAX_NB_CITIES + 1];

	if(idx >= nb_perm || cont_tasks == tasks_per_thread) return INT_MAX;

	// Get and calculate permutations for the current thread
	while (idx < nb_perm && cont_tasks != tasks_per_thread) {

		// compute factorial code
		for (int k = 0; k < n; ++k) {
			perm[k] = idx / fact[n - 1 - k];
			idx = idx % fact[n - 1 - k];
		}

		// readjust values to obtain the permutation
		for (int k = n - 1; k > 0; --k) {
			for (int j = k - 1; j >= 0; --j) {
				if (perm[j] <= perm[k]) {
					perm[k]++;
				}
			}
		}

		// Loop path
		perm[n] = perm[0];

		// Perm cost
		cost = 0;

		// Calc perm cost
		for (int i = 0; i < n; i++) {
			cost += dist[perm[i] * n + perm[i + 1]];
		}

		idx = idx + limit_blocks_1d;
		cont_tasks++;
		
		// maintain min cost
		cost = cost < min_cost ? cost : min_cost;
	}

	return cost;
}

__global__ void perm_cuda(int n, ll nb_perm, int * dist, ll * fact, int * mcost, int tasks_per_thread, ll limit_blocks_1d) {
	// Thread index
	int tidx = threadIdx.x;

	// Block index
	int bidx = blockIdx.x;

	// Global index
	ll idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Shared dist matrix
	__shared__ int s_dist[MAX_NB_CITIES*MAX_NB_CITIES];
	
	// Copy global dist matrix to shared dist matrix
	int step = (n*n + blockDim.x - 1)/(blockDim.x);
	int start = tidx*step;
	for (int i = 0; i < step; i++) {
		if ((start + i) < (n*n)) {
			s_dist[start + i] = dist[start + i];
		}
	}

	// Shared factorial
	__shared__ ll s_fact[MAX_NB_CITIES];

	// Copy global factorial to shared factorial
	if (tidx < n) {
		s_fact[tidx] = fact[tidx];
	}
	__syncthreads();

	// Shared cost
	__shared__ int s_mcost[MAX_THREAD_PER_BLOCK];

	// Minimal local cost
	s_mcost[tidx] = calc_perm_cost(idx, n, nb_perm, s_dist, s_fact, tasks_per_thread, limit_blocks_1d);
	__syncthreads();

	// Reduce local cost to find global
	ll step_size = 1;
	int nb_threads = blockDim.x/2;

	while (nb_threads > 0) {
		if (tidx < nb_threads) {
			int fst = tidx * step_size * 2;
			int snd = fst  + step_size;

			if (s_mcost[snd] < s_mcost[fst]) {
				s_mcost[fst] = s_mcost[snd];
			}
		}

		step_size <<= 1;
		nb_threads >>= 1;

		__syncthreads();
	}

	// Put mcost from block in global
	if (tidx == 0) {
		mcost[bidx] = s_mcost[0];
	}
}

void printDeviceProps() {
	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	printf("prop.name                        %s\n",       prop.name);
	printf("prop.totalGlobalMem              %zdGB\n",    prop.totalGlobalMem >> 10 >> 10 >> 10);
	printf("prop.sharedMemPerBlock           %zdKB\n",    prop.sharedMemPerBlock >> 10);
	printf("prop.regsPerBlock                %d\n",       prop.regsPerBlock);
	printf("prop.warpSize                    %d\n",       prop.warpSize);
	printf("prop.memPitch                    %zdGB\n",    prop.memPitch >> 10 >> 10 >> 10);
	printf("prop.maxThreadsPerBlock          %d\n",       prop.maxThreadsPerBlock);
	printf("prop.maxThreadsDim               %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("prop.maxGridSize                 %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("prop.totalConstMem               %zdKB\n",    prop.totalConstMem >> 10);
	printf("prop.multiProcessorCount         %d\n",       prop.multiProcessorCount);
	printf("prop.l2CacheSize                 %dMB\n",     prop.l2CacheSize >> 10 >> 10);
	printf("prop.maxThreadsPerMultiProcessor %d\n",       prop.maxThreadsPerMultiProcessor);
	printf("prop.sharedMemPerMultiprocessor  %zdKB\n",    prop.sharedMemPerMultiprocessor >> 10);
	printf("prop.regsPerMultiprocessor       %d\n",       prop.regsPerMultiprocessor);
	printf("\n");
}

int * putDMatrixInDevice(int n) {
	// X coords
	int * x = (int *)malloc(n*sizeof(int));
	if (!x) {
		printf("malloc error\n");
		exit(1);
	}

	// Y coords
	int * y = (int *)malloc(n*sizeof(int));
	if (!y) {
		printf("malloc error\n");
		exit(1);
	}

	// Read
	for (int i = 0; i < n; i++) {
		scanf("%d %d", &x[i], &y[i]);
	}

	// Dist matrix on host
	int * h_dist = (int *)malloc(n*n*sizeof(int));
	if (!h_dist) {
		printf("malloc error\n");
		exit(1);
	}

	// Calc dist matrix
	int dx, dy;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			dx = x[i] - x[j];
			dy = y[i] - y[j];
			h_dist[i*n + j] = (int)sqrt(dx*dx + dy*dy);
		}
	}

	free(x);
	free(y);

	#ifdef DEBUG
		// Print dist
		printf("dist\n");
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				printf("%3d ", h_dist[i*n + j]);
			}
			printf("\n");
		}
	#endif

	// Dist matrix on device
	int * d_dist;
	if (hipMalloc(&d_dist, n*n*sizeof(int)) == hipErrorOutOfMemory) {
		printf("hipMalloc error\n");
		exit(1);
	}

	// Copy host to device
	hipMemcpy(d_dist, h_dist, n*n*sizeof(int), hipMemcpyHostToDevice);

	free(h_dist);

	return d_dist;
}

ll * putFactInDevice(int n) {
	// Fact vector on host
	ll * h_fact = (ll *)malloc(n*sizeof(ll));
	if (!h_fact) {
		printf("malloc error\n");
		exit(1);
	}

	// Calc fact
	h_fact[0] = 1;
	for (int i = 1; i < n; i++) {
		h_fact[i] = i * h_fact[i - 1];
	}

	#ifdef DEBUG
		// Print fact
		printf("fact ");
		for (int i = 0; i < n; i++) {
			printf("%lld ", h_fact[i]);
		}
		printf("\n");
	#endif

	// Fact vector on device
	ll * d_fact;
	if (hipMalloc(&d_fact, n*sizeof(ll)) == hipErrorOutOfMemory) {
		printf("hipMalloc error\n");
		exit(1);
	}

	// Copy host to device
	hipMemcpy(d_fact, h_fact, n*sizeof(ll), hipMemcpyHostToDevice);

	return d_fact;
}

ll factorial(int n) {
	return (n > 1 ? n*factorial(n-1) : 1);
}

int run_tsp() {
	int nb_cities, tasks_per_thread = 1;
	scanf("%d", &nb_cities);

	// Dist matrix
	int * d_dist = putDMatrixInDevice(nb_cities);

	// Fact vector
	ll * d_fact = putFactInDevice(nb_cities);

	// Number of permutations
	ll nb_perm = factorial(nb_cities);

	#ifdef DEBUG
		printf("nb_cities\t\t%d\n", nb_cities);
		printf("nb_perm\t\t\t%lld\n", nb_perm);
	#endif

	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	// Calculate number of tasks per thread
	ll limit_blocks_1d = prop.maxGridSize[0];
	if(nb_perm / MAX_THREAD_PER_BLOCK > limit_blocks_1d) {
		tasks_per_thread = ceil((nb_perm / MAX_THREAD_PER_BLOCK) / limit_blocks_1d);
	}

	// Set number of tasks/thread and # blocks
	ll nb_blocks, nb_threads;
	if(nb_perm > limit_blocks_1d) {
		nb_blocks = limit_blocks_1d;
		nb_threads = MAX_THREAD_PER_BLOCK;
	} else {
		nb_blocks  = (nb_perm + MAX_THREAD_PER_BLOCK - 1)/MAX_THREAD_PER_BLOCK;
		nb_threads = (nb_perm > MAX_THREAD_PER_BLOCK ? MAX_THREAD_PER_BLOCK : nb_perm);
	}

	#ifdef DEBUG
		printf("nb_blocks\t\t%lld\n", nb_blocks);
		printf("nb_threads\t\t%lld\n", nb_threads);
		printf("tasks/thread\t\t%lld\n", tasks_per_thread);
	#endif

	// Mim cost from blocks
	int * h_mcost = (int *) malloc(nb_blocks*sizeof(int));
	int * d_mcost;
	hipMalloc(&d_mcost, nb_blocks*sizeof(int));

	// Call gpu
	clock_t start = clock();
	perm_cuda<<<nb_blocks, nb_threads>>>(nb_cities, nb_perm, d_dist, d_fact, d_mcost, tasks_per_thread, limit_blocks_1d);
	hipDeviceSynchronize();
	clock_t end = clock();

	// Time
	int msec = (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("%ds %dms\n", msec/1000, msec%1000);

	// Copy device to host, ret
	hipMemcpy(h_mcost, d_mcost, nb_blocks*sizeof(int), hipMemcpyDeviceToHost);

	// Error check
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA Error: %s\n",hipGetErrorString(err));
	}

	// Host global min
	int global_mcost = INT_MAX;

	// Find global min
	for (int i = 0; i < nb_blocks; i++) {
		if (h_mcost[i] < global_mcost) {
			global_mcost = h_mcost[i];
		}
	}

	hipFree(d_fact);
	hipFree(d_dist);
	hipFree(d_mcost);

	return global_mcost;
}

int main() {
	#ifdef DEBUG
		printDeviceProps();
	#endif

	int num_instances;
	
	scanf("%d", &num_instances);

	for (int i = 0; i < num_instances; i++) {
		printf("%d\n", run_tsp());
	}

	return 0;
}